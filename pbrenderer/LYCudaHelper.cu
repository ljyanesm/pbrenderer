#include "LYCudaHelper.cuh"

void LYCudaHelper::allocateHostArray(void **devPtr, size_t size, unsigned int flag)
{
	checkCudaErrors(hipHostAlloc(devPtr, size, hipHostMallocMapped));
}

void LYCudaHelper::memsetDeviceArray(void **devPtr, size_t size, int value)
{
	checkCudaErrors(hipMemset(devPtr, value, size));
}

void LYCudaHelper::freeHostArray(void *devPtr)
{
	checkCudaErrors(hipHostFree(devPtr));
}

void LYCudaHelper::allocateArray(void **devPtr, size_t size)
{
	checkCudaErrors(hipMalloc(devPtr, size));
}

void LYCudaHelper::freeArray(void *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}

void LYCudaHelper::registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
		cudaGraphicsMapFlagsNone));
}

void LYCudaHelper::unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
}

void *LYCudaHelper::mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
	void *ptr;
	checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
		*cuda_vbo_resource));
	return ptr;
}

void LYCudaHelper::unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void LYCudaHelper::copyArrayToDevice(void *device, const void *host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void LYCudaHelper::copyArrayFromDevice(void *host, const void *device,
struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	if (cuda_vbo_resource)
	{
		device = mapGLBufferObject(cuda_vbo_resource);
	}

	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

	if (cuda_vbo_resource)
	{
		unmapGLBufferObject(*cuda_vbo_resource);
	}
}

void LYCudaHelper::getMappedPointer(void **device, void *host, uint flag)
{
	checkCudaErrors(hipHostGetDevicePointer(device, host, flag));
}

void LYCudaHelper::printMemInfo()
{
	size_t gpuFreeMem, gpuTotalMem;
	hipError_t error(hipMemGetInfo(&gpuFreeMem, &gpuTotalMem));
	std::cout << "hipMemGetInfo error code (" << error << "): " << hipGetErrorString(error) << std::endl;
	gpuFreeMem /= 1024*1024;
	gpuTotalMem /= 1024*1024;
	printf("Total amount of MB available: %Iu MB \nTotal amount of device memory: %Iu MB\n", gpuFreeMem, gpuTotalMem);
}