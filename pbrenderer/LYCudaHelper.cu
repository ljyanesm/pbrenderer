#include "LYCudaHelper.cuh"

void LYCudaHelper::allocateArray(void **devPtr, int size)
{
	checkCudaErrors(hipMalloc(devPtr, size));
}

void LYCudaHelper::freeArray(void *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}

void LYCudaHelper::registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
		cudaGraphicsMapFlagsNone));
}

void LYCudaHelper::unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
}

void *LYCudaHelper::mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
	void *ptr;
	checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
		*cuda_vbo_resource));
	return ptr;
}

void LYCudaHelper::unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void LYCudaHelper::copyArrayToDevice(void *device, const void *host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void LYCudaHelper::copyArrayFromDevice(void *host, const void *device,
struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	if (cuda_vbo_resource)
	{
		device = mapGLBufferObject(cuda_vbo_resource);
	}

	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

	if (cuda_vbo_resource)
	{
		unmapGLBufferObject(*cuda_vbo_resource);
	}
}

