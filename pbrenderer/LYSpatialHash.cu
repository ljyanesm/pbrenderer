#include "hip/hip_runtime.h"
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <vector_functions.h>
#include <hip/device_functions.h>


#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>


#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "LYSpatialHash_impl.cuh"

extern "C" {

	void setParameters(SimParams *hostParams)
	{
		// copy parameters to constant memory
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
	}


	//Round a / b to nearest higher integer value
	uint iDivUp(uint a, uint b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	// compute grid and thread block size for a given number of elements
	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void calcHash(uint  *gridParticleHash,
		uint  *gridParticleIndex,
		LYVertex *pos,
		int    numVertices)
	{
		uint numThreads, numBlocks;
		computeGridSize(numVertices, 256, numBlocks, numThreads);

		// execute the kernel
		calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
			gridParticleIndex,
			(LYVertex *) pos,
			numVertices);

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed");
	}

	void reorderDataAndFindCellStart(uint  *cellStart,
		uint  *cellEnd,
		LYVertex *sortedPos,
		uint  *gridParticleHash,
		uint  *gridParticleIndex,
		LYVertex *oldPos,
		uint   numVertices,
		uint   numCells)
	{
		uint numThreads, numBlocks;
		computeGridSize(numVertices, 256, numBlocks, numThreads);
		// set all cells to empty
		checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numVertices*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numVertices*sizeof(float4)));
#endif

		uint smemSize = sizeof(uint)*(numThreads+1);
		reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
			cellStart,
			cellEnd,
			(LYVertex *) sortedPos,
			gridParticleHash,
			gridParticleIndex,
			(LYVertex *) oldPos,
			numVertices);
		getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
		checkCudaErrors(hipUnbindTexture(oldPosTex));
		checkCudaErrors(hipUnbindTexture(oldVelTex));
#endif
	}

	void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numVertices)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
			thrust::device_ptr<uint>(dGridParticleHash + numVertices),
			thrust::device_ptr<uint>(dGridParticleIndex));
	}

	void collisionCheck(float3 pos, LYVertex *sortedPos, uint *gridParticleIndex, uint *cellStart, uint *cellEnd, float3 *forceFeedback, uint numVertices)
	{
#if USE_TEX
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
#endif

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numVertices, 64, numBlocks, numThreads);
		
		// execute the kernel
        collisionCheckD<<< numBlocks, numThreads >>>(pos,
											(LYVertex *)sortedPos,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
											  forceFeedback,
                                              numVertices);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(oldVelTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
	}
	
}